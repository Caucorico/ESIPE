#include "hip/hip_runtime.h"
#include "SDL_image.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>
#include "cuStopwatch.cu"

#define OFFSET 3

enum conv_t{
    CONV_IDENTITY,
    CONV_EDGE,
    CONV_SHARP,
    CONV_GAUSS
};

SDL_Window *screen;
SDL_Window *screen_res;
SDL_Renderer *ren;
SDL_Renderer *ren_res;
SDL_Texture *tex;
SDL_Texture *tex_res;
SDL_Surface *surf;

int32_t width, height;
float filter[9];

float __constant__ filter_device[9];

__global__ void conv_global(const unsigned char* src, unsigned char* dest, int32_t w, int32_t h){
    // todo: write a kernel to apply the given filter on the given image stored in the global memory
    int32_t i, j, avg_r, avg_g, avg_b;
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t buff, buff2;

    while ( tid < w*h ) {
        avg_r = avg_g = avg_b = 0;

        i = tid-w;
        buff2 = 0;
        while ( i <= tid+w ) {
            for ( j = -1 ; j <= 1 ; j++ ) {
                buff = (i+j)*3;
                avg_r += src[buff+0]*filter_device[buff2];
                avg_g += src[buff+1]*filter_device[buff2];
                avg_b += src[buff+2]*filter_device[buff2];
                buff2++;
            }

            i += w;
        }


        dest[tid*3] = avg_r < 0 ? 0 : avg_r > 255 ? 255 : avg_r;
        dest[tid*3+1] = avg_g < 0 ? 0 : avg_g > 255 ? 255 : avg_g;
        dest[tid*3+2] = avg_b < 0 ? 0 : avg_b > 255 ? 255 : avg_b;
        tid += gridDim.x;
    }

}

float conv_global_gpu(unsigned char* pixels, int32_t w, int32_t h){
    // todo: write the code that manages memory (global memory) and invokes the kernel conv_global, it should return the running time
    unsigned char* buff, *dest;
    int32_t total_size;
    cuStopwatch sw;


    sw.start();

    total_size = w*h*3;
    hipMalloc(&buff, sizeof(unsigned char) * total_size);
    hipMalloc(&dest, sizeof(unsigned char) * total_size);

    hipMemcpy(buff, pixels, total_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    conv_global<<<(1<<10),(1<<10)>>>(buff, dest, w, h);
    hipMemcpy(pixels, dest, total_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(buff);
    hipFree(dest);

    return sw.stop();
}

__global__ void conv_texture(hipTextureObject_t src, unsigned char* dest, int32_t w, int32_t h){
    // todo: write a kernel to apply the given filter on the given image stored as a texture
}

float conv_texture_gpu(unsigned char* pixels, int32_t w, int32_t h){
    // todo: write the code that manages memory (texture memory) and invokes the kernel conv_global, it should return the running time
    return 0.0;
}

int main(int argc, char** argv){
    SDL_Event event;
    bool withtex = false;
    
    // Initialize SDL
    if( SDL_Init(SDL_INIT_VIDEO) < 0 ) {
        fprintf(stderr, "Couldn't initialize SDL: %s\n", SDL_GetError());
        exit(1);
    }
	atexit(SDL_Quit);
    
    if(argc == 1){
        exit(1);
    }
    
    // Read image and option
    IMG_Init(IMG_INIT_PNG);
    surf = IMG_Load(argv[1]);
    if(surf == NULL){
        fprintf(stderr, "Error loading image.\n");
        exit(1);
    }
    width = surf->w;
    height = surf->h;
    SDL_SetSurfaceRLE(surf, 1);
    
    // Initialize involution kernel
    conv_t conv_type;
    if(argc >= 3){
        if (strcmp(argv[2], "identity") == 0) conv_type = CONV_IDENTITY;
        else if (strcmp(argv[2], "edge") == 0) conv_type= CONV_EDGE;
        else if (strcmp(argv[2], "sharp") == 0) conv_type= CONV_SHARP;
        else if (strcmp(argv[2], "gauss") == 0) conv_type = CONV_GAUSS;
        else conv_type = CONV_IDENTITY;
    }
    switch(conv_type){
        case CONV_EDGE:
            filter[0] = -1; filter[1] = -1; filter[2] = -1; 
            filter[3] = -1; filter[4] = 8; filter[5] = -1; 
            filter[6] = -1; filter[7] = -1; filter[8] = -1; 
            break;
        case CONV_SHARP:
            filter[0] = 0; filter[1] = -1; filter[2] = 0; 
            filter[3] = -1; filter[4] = 5; filter[5] = -1; 
            filter[6] = 0; filter[7] = -1; filter[8] = 0; 
            break;
        case CONV_GAUSS:
            filter[0] = 1.0f/16; filter[1] = 1.0f/8; filter[2] = 1.0f/16; 
            filter[3] = 1.0f/8; filter[4] = 1.0f/4; filter[5] = 1.0f/8; 
            filter[6] = 1.0f/16; filter[7] = 1.0f/8; filter[8] = 1.0f/8; 
            break;
        default:
            filter[0] = 0; filter[1] = 0; filter[2] = 0; 
            filter[3] = 0; filter[4] = 1; filter[5] = 0; 
            filter[6] = 0; filter[7] = 0; filter[8] = 0; 
            break;
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(filter_device), filter, sizeof(float)*9, 0, hipMemcpyHostToDevice);
    
    if(argc >= 4){
        if(strcmp(argv[3], "texture") == 0) withtex = true;
    }
    
    // Create window
	screen = SDL_CreateWindow("Original", 
                        100,
                        100,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    screen_res = SDL_CreateWindow("Filtered", 
                        300,
                        300,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen_res == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    
    // Initialize CUDA
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    
    // Create renderer and texture
    ren = SDL_CreateRenderer(screen, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex = SDL_CreateTextureFromSurface(ren, surf);
    
    // Show image
    SDL_RenderCopy(ren, tex, NULL, NULL);
    SDL_RenderPresent(ren);
    
    // Compute
    SDL_LockSurface(surf);
    float elapsed;
    if(withtex){
        elapsed = conv_texture_gpu((unsigned char*)surf->pixels, width, height);
    }else{
        elapsed = conv_global_gpu((unsigned char*)surf->pixels, width, height);
    }
    SDL_UnlockSurface(surf);
    
    // Show computed image
    ren_res = SDL_CreateRenderer(screen_res, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex_res = SDL_CreateTextureFromSurface(ren_res, surf);
    SDL_RenderCopy(ren_res, tex_res, NULL, NULL);
    SDL_RenderPresent(ren_res);
    SDL_FreeSurface(surf);
    
    while (1) {
        SDL_WaitEvent(&event);
        if ((event.type == SDL_QUIT) || ((event.type == SDL_WINDOWEVENT) && (event.window.event == SDL_WINDOWEVENT_CLOSE))) break;
    }
    
    char s[100];
    sprintf(s, "Kernel execution time: %.4fms", elapsed);
    SDL_ShowSimpleMessageBox(SDL_MESSAGEBOX_INFORMATION, "Timing", s, screen);
    SDL_DestroyTexture(tex);
    SDL_DestroyRenderer(ren);
    SDL_DestroyWindow(screen);
    SDL_DestroyWindow(screen_res);
    exit(0);
}