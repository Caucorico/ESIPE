#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <time.h>
#include <math.h>
#include "cuStopwatch.cu"

#define COUNT (1<<23)
#define LINK_PER_PAGE 4
#define ERMIX 0.25f
#define MAXINT (4294967295.0f)
#define DAMPING 0.9f
#define EPSILON 0.00000001f
#define MAXPRCOUNT 16
#define INITPROJ 1024

/* ------------ Pagerank computation, GPU part ------------ */

__global__ void pr_init_gpu(float* pr){
    // TODO: fill in initial value for pagerank
}

__global__ void pr_damping_gpu(float* pr){
    // TODO: fill in (1 - damping constant) for pagerank
}

__global__ void pr_iter_gpu(const uint2* links, const float* oldp, float* newp){
    // TODO: add contributions for each link for pagerank
}

__global__ void pr_conv_check_gpu(const float* oldp, const float* newp, uint32_t* conv){
    // TODO: check for convergence against 
}

float pr_compute_gpu(const uint2* links, float* pr){
    // TODO: control GPU computation, returns computation time (in seconds, not counting memory transfer time)
}

/* ------------ Pagerank computation, CPU part ------------ */

__global__ void pr_init_cpu(float* pr){
    // TODO: equivalence of pr_init_gpu on host
}

__global__ void pr_damping_cpu(float* pr){
    // TODO: equivalence of pr_damping_gpu on host
}

void pr_iter_cpu(const uint2* links, const float* oldp, float* newp){
    // TODO: equivalenc of pr_iter_gpu on host
}

void pr_conv_check_cpu(const float* oldp, const float* newp, uint32_t* conv){
    // TODO: equivalence of pr_conv_check_gpu on host
}

float pr_compute_cpu(const uint2* links, float* pr){
    // TODO: equivalence of pr_compute_gpu on host
}

/* ------------ Random graph generation ------------ */

uint32_t randstate;

uint32_t myrand(){
    randstate ^= randstate << 13;
    randstate ^= randstate >> 17;
    randstate ^= randstate << 5;
    return randstate;
}

void seed(){
    randstate = time(NULL);
    for(int i = 0; i < 16; i++) myrand();
    return;
}

void randgen(uint2* links){
    uint32_t state = time(NULL);
    uint32_t *weight = (uint32_t*)malloc(sizeof(uint32_t) * COUNT);
    memset((void*)weight, 0, sizeof(uint32_t) * COUNT);
    uint32_t totalweight = 0;
    uint32_t lcnt = 0;
    
    // Initial five
    for(int i = 0; i < INITPROJ; i++){
        weight[i] = 1;
        for(int j = 0; j < 4; j++){
            links[lcnt].x = i;
            links[lcnt].y = (uint32_t)(myrand()*(COUNT/MAXINT));
            lcnt++;
        }
    }
    totalweight = INITPROJ;
    
    // Barabasi-Albert with Erdos-Renyi mix-in
    for(uint32_t i = INITPROJ; i < COUNT; i++){
        for(int k = 0; k < LINK_PER_PAGE; k++){
             if(myrand()/MAXINT < ERMIX){
                links[lcnt].x = i;
                links[lcnt].y = (uint32_t)(myrand()*(COUNT/MAXINT));
                lcnt++;
            }else{
                uint32_t randweight = (uint32_t)(myrand()/MAXINT*totalweight);
                uint32_t idx = 0;
                while(randweight > weight[idx]){
                    randweight -= weight[idx];
                    idx++;
                }
                links[lcnt].x = i;
                links[lcnt].y = idx;
                lcnt++;
                weight[idx]++;
                totalweight++;
            }
        }
    }
    return;
}

/* ------------ Main control ------------ */

void pr_extract_max(const float* pr, float* prmax, uint32_t* prmaxidx){
    for(int i = 0; i < MAXPRCOUNT; i++) prmax[i] = -1.0f;
    for(uint32_t i = 0; i < COUNT; i++){
        if(pr[i] > prmax[MAXPRCOUNT-1]){
            int ptr = 0;
            while(pr[i] <= prmax[ptr]) ptr++;
            float oldval, newval;
            uint32_t oldidx, newidx;
            newval = pr[i];
            newidx = i;
            for(int j = ptr; j < MAXPRCOUNT; j++){
                oldval = prmax[j];
                oldidx = prmaxidx[j];
                prmax[j] = newval;
                prmaxidx[j] = newidx;
                newval = oldval;
                newidx = oldidx;
            }
        }
    }
    return;
}

int main(){
    // Generating random network
    uint2* randlinks;
    hipHostAlloc((void**)&randlinks, sizeof(uint2)*COUNT*LINK_PER_PAGE, hipHostMallocDefault);
    seed();
    randgen(randlinks);
    printf("Finished generating graph\n\n");
    
    // Declaration of needed variables and arrays
    float prmax[MAXPRCOUNT];
    uint32_t prmaxidx[MAXPRCOUNT];
    float elapsed;
    float *pagerank;
    float check;
    hipHostAlloc((void**)&pagerank, sizeof(float)*COUNT, hipHostMallocDefault);
    
    // Processing by GPU
    elapsed = pr_compute_gpu(randlinks, pagerank);
    printf("GPU version, runtime %7.4fs\n", elapsed);
    check = 0.0f;
    for(uint32_t i = 0; i <COUNT; i++) check+=pagerank[i];
    printf("Deviation: %.6f\n", check);
    pr_extract_max(pagerank, prmax, prmaxidx);
    for(int i = 0; i < MAXPRCOUNT; i++){
        printf("Rank %d, index %u, normalized pagerank %8.7f\n", i, prmaxidx[i], prmax[i] / check);
    }
    printf("\n");
    
    // Processing by CPU
    elapsed = pr_compute_cpu(randlinks, pagerank);
    printf("CPU version, runtime %7.4fs\n", elapsed);
    check = 0.0f;
    for(uint32_t i = 0; i <COUNT; i++) check+=pagerank[i];
    printf("Deviation: %.6f\n", check);
    pr_extract_max(pagerank, prmax, prmaxidx);
    for(int i = 0; i < MAXPRCOUNT; i++){
        printf("Rank %d, index %u, normalized pagerank %8.7f\n", i, prmaxidx[i], prmax[i] / check);
    }
    
    // Free memory
    hipHostFree(randlinks);
    hipHostFree(pagerank);
	return 0;
}