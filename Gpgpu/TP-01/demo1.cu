#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel() {
	printf("%d, %d\n", threadIdx.x, blockIdx.x);
	return;
}

int main() {
	// main iteration
	kernel <<<16, 4, 0>>>();
	return 0;
}