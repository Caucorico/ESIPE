#include <hip/hip_runtime.h>


class cuStopwatch{
	// todo: add your internal data structure, all in private
	private:
		hipEvent_t start_event;
		hipEvent_t end_event;
		bool is_watching;

	public:
		cuStopwatch();
		~cuStopwatch();
		void start();
		float stop();
};

cuStopwatch::cuStopwatch(){
	hipEventCreate(&start_event);
	hipEventCreate(&end_event);
}

cuStopwatch::~cuStopwatch(){
	hipEventDestroy(start_event);
	hipEventDestroy(end_event);
}

void cuStopwatch::start(){
	hipEventRecord(start_event);
}

float cuStopwatch::stop(){
	float elapsed_time;
	hipEventRecord(end_event);
	hipEventSynchronize(end_event);
	hipEventElapsedTime(&elapsed_time, start_event, end_event);
	return elapsed_time;
}
