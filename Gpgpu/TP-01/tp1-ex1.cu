#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
	hipDeviceProp_t* cdp = (hipDeviceProp_t*) malloc(sizeof(hipDeviceProp_t));
	int deviceCount = 0, i;
	hipGetDeviceCount(&deviceCount);
	printf("Number of devices : %d\n", deviceCount);

	for ( i = 0 ; i < deviceCount ; i++ )
	{
		hipGetDeviceProperties(cdp, i);
		printf("Device n°%d (%s)\n", i, cdp->name);
		printf("frequency : %d KHz\n", cdp->clockRate);
		printf("Global memory size : %zd bytes\n", cdp->totalGlobalMem);
		printf("WarpSize : %d threads\n", cdp->warpSize);
	}

	free(cdp);
	return 0;
}
