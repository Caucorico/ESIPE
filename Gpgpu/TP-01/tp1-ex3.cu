#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void kernel() {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t n = tid;
	uint32_t sum = 0;
    uint32_t prod = 1;
    while(n != 0){
        uint32_t digit = n % 10;
        n /= 10;
        sum += digit;
        prod *= digit;
    }
    if(sum*prod == tid) printf("%u\n", tid);
	return;
}

void checkrange(uint32_t range){
    double dim = sqrt(range);
	uint32_t thread_number = (uint32_t)ceil(range/(dim));

	if ( thread_number > 1024 ) {
		printf("Impossible to run more threads than 1024.\nNumber reduce to 1024. \n");
		thread_number = 1024;
	}

    printf("Checking %u for sum-product numbers\n", range);
    /* If the number of threads is greater than 1024, the code will not be executed. */
    kernel<<<(uint32_t)dim, thread_number, 0>>>();
    hipDeviceSynchronize();
}

int main() {
	// main iteration
	checkrange(1024);
    checkrange(16777216);
	return 0;
}